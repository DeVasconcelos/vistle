#include "hip/hip_runtime.h"
/**\file
 * \brief class ReadBackCuda
 * 
 * \author Martin Aumüller <aumueller@hlrs.de>
 * \author Martin Aumüller <aumueller@uni-koeln.de>
 * \author (c) 2011 RRZK, 2013 HLRS
 *
 * \copyright GPL2+
 */

#include <GL/glew.h>
#include "ReadBackCuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <cstdio>
#include <cassert>

#include "rfbext.h"

//! use BGRA for pixel transfer
#define USE_BGRA

static const int TileSizeX = 16;
static const int TileSizeY = 16;

bool ReadBackCuda::s_error = false;

//! check for CUDA errors
bool cucheck(const char *msg, hipError_t err)
{
#if 0
   if(err == hipSuccess)
   {
      hipDeviceSynchronize();
      err = hipGetLastError();
   }
#endif

   if(err == hipSuccess)
      return true;

   fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));

   ReadBackCuda::s_error = true;

   return false;
}

//! ctor
ReadBackCuda::ReadBackCuda()
: pboName(0)
, imgRes(NULL)
, outImg(NULL)
, imgSize(0)
, compSize(0)
, m_initialized(false)
{
}

//! GL initialization, requires valid GL context
bool ReadBackCuda::init()
{
   if (s_error)
      return false;

   assert(m_initialized == false);

   GLenum err = glewInit();
   if (GLEW_OK != err)
   {
      fprintf(stderr, "glewInit failed: %s\n", glewGetErrorString(err));
      return false;
   }

   bool canUsePbo = glGenBuffers && glDeleteBuffers && glBufferData && glBindBuffer;
   if(!canUsePbo)
   {
      fprintf(stderr, "PBOs not available\n");
      return false;
   }

   m_initialized = true;

   return true;
}

//! query initialization state
bool ReadBackCuda::isInitialized() const {

   return m_initialized && !s_error;
}

//! dtor
ReadBackCuda::~ReadBackCuda()
{
   if(imgRes)
      cucheck("unreg buf", hipGraphicsUnregisterResource(imgRes));
   cucheck("cufree out", hipFree(outImg));
   if(pboName != 0)
      glDeleteBuffers(1, &pboName);
}

//! set up PBO for image size to be read back
bool ReadBackCuda::initPbo(size_t raw, size_t compressed)
{
   if (s_error)
      return false;

   if(imgSize != raw || compSize != compressed || pboName == 0)
   {
      if(pboName == 0)
      {
         glGenBuffers(1, &pboName);
      }
      else
      {
         cucheck("unreg buf", hipGraphicsUnregisterResource(imgRes));
         cucheck("cufree out", hipFree(outImg));
      }

      glBindBuffer(GL_PIXEL_PACK_BUFFER, pboName);
      glBufferData(GL_PIXEL_PACK_BUFFER, raw, NULL, GL_STREAM_COPY);
      glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);

      imgSize = raw;

      cucheck("reg buf", hipGraphicsGLRegisterBuffer(&imgRes, pboName, cudaGraphicsMapFlagsNone));
      cucheck("malloc out", hipMalloc(&outImg, compressed));
   }

   return pboName != 0;
}

//! convert single color from RGB to YUV
__device__ void colorconvert(uchar r, uchar g, uchar b, uchar *y, uchar *u, uchar *v)
{
   *y = 0.299f*r + 0.587f*g + 0.114f*b;
   *u = 128.f + 0.5f*r - 0.418688f*g - 0.081312f*b;
   *v = 128.f - 0.168736f*r - 0.331265f*g + 0.5f*b;
}

//! convert single color from RGB to YUV
__device__ void colorconvert(uchar r, uchar g, uchar b, uchar *y, float *u, float *v)
{
   *y = 0.299f*r + 0.587f*g + 0.114f*b;
   *u += 128.f + 0.5f*r - 0.418688f*g - 0.081312f*b;
   *v += 128.f - 0.168736f*r - 0.331265f*g + 0.5f*b;
}

//! convert buffer from RGB to YUV
__global__ void rgb2yuv(const uchar *inimg, uchar *outimg, int w, int h, int subx, int suby)
{
   if(subx<1) subx=1;
   if(suby<1) suby=1;
#ifdef USE_BGRA
   const int bpp = 4;
#else
   const int bpp = 3;
#endif

   const int wsub = (w+subx-1)/subx;
   const int hsub = (h+suby-1)/suby;

   uchar *ybase = outimg;
   uchar *ubase = &ybase[w*h];

   int x=threadIdx.x+blockIdx.x*TileSizeX;
   int y=threadIdx.y+blockIdx.y*TileSizeY;

   uchar *vbase = &ubase[wsub*hsub];

   float u=0.f, v=0.f;

   uchar *uu = &ubase[x+y*wsub];
   uchar *vv = &vbase[x+y*wsub];

   x *= subx;
   y *= suby;

   if(x>=w || y>=h)
      return;

   for(int iy=0; iy<suby; ++iy)
   {
      const int yy=y+iy;
      for(int ix=0; ix<subx; ++ix)
      {
         const int xx=x+ix;
         uchar *cy = &ybase[xx+yy*w];
         const uchar r = inimg[(xx+yy*w)*bpp+0];
         const uchar g = inimg[(xx+yy*w)*bpp+1];
         const uchar b = inimg[(xx+yy*w)*bpp+2];

         colorconvert(r,g,b, cy,&u,&v);
      }
   }
   *uu = u/(subx*suby);
   *vv = v/(subx*suby);
}

template<int nelem>
__device__ void findgap(uint32_t * __restrict__ data,
      uint32_t * __restrict__ order,
      uint32_t maxidx,
      uint32_t midval,
      uint32_t & __restrict__ mididx) {

   if (threadIdx.x == nelem-1) {
      mididx = maxidx/2;
   }
   if (threadIdx.x+1 < maxidx) {
      if (data[order[threadIdx.x]] <= midval
            && data[order[threadIdx.x+1]] > midval)
         mididx = threadIdx.x;
   }
}

template<int nelem>
__device__ void sort(uint32_t * __restrict__ data,
      uint32_t * __restrict__ ranks,
      uint32_t * __restrict__ order,
      uint32_t Far,
      uint32_t & __restrict__ maxidx) {

   const int halfwarp = 16;
#define SYNC() if (nelem > halfwarp) __syncthreads()

   int rank = 0;
   for (int i=0; i<nelem; ++i) {
      rank += data[i]<data[threadIdx.x];
   }
   ranks[threadIdx.x] = rank;
   SYNC();

   if (nelem > halfwarp) {
      int add = 0;
      for (int i=0; i<nelem-1; ++i) {
         if (threadIdx.x > i)
            add += ranks[threadIdx.x]==ranks[i];
      }
      SYNC();
      ranks[threadIdx.x] += add;
   } else {
      for (int i=0; i<nelem-1; ++i) {
         if (threadIdx.x > i)
            ranks[threadIdx.x] += ranks[threadIdx.x]==ranks[i];
      }
   }

   order[ranks[threadIdx.x]] = threadIdx.x;
   SYNC();

   if (threadIdx.x == 0)
      maxidx = nelem-1;
   SYNC();
   if (threadIdx.x > 0) {
      if (data[order[threadIdx.x]] == Far
            && data[order[threadIdx.x-1]] < Far)
         maxidx = threadIdx.x-1;
   }
   SYNC();

#undef SYNC
}

//! dxt-like compresesion for depth values
template<int bpp, class Quant, bool RGBA>
__global__ void depthquant(const uchar *inimg, uchar *outimg, int w, int h)
{
   const int edge = Quant::edge;
   const int scalebits = Quant::scale_bits;
   const int quantbits = Quant::bits_per_pixel;
   const uint32_t Far = bpp==4 ? 0x00ffffffU : (1U<<(bpp*8))-1;
   const uint32_t mask = (1U << quantbits)-1;

   int x = (threadIdx.x%edge) + (threadIdx.y%(TileSizeX/edge))*edge + blockIdx.x*TileSizeX;
   if (x >= w)
      x = w-1;
   int y = (threadIdx.x/edge) + (threadIdx.y/(TileSizeY/edge))*edge + blockIdx.y*TileSizeY;
   if (y >= h)
      y = h-1;

   extern __shared__ uint32_t shared[];
   const int tileoff = blockDim.x*threadIdx.y;
   uint32_t *depths = &shared[tileoff+0];
   uint32_t *ranks = &depths[blockDim.x*blockDim.y];
   uint32_t *orders = &ranks[blockDim.x*blockDim.y];
   uint32_t *maxidx = &shared[blockDim.x*blockDim.y*3+threadIdx.y];

   const uint32_t depth = get_depth<RGBA?DepthRGBA:DepthInteger, bpp>(inimg, x, y, w, h);

   orders[threadIdx.x] = 0;
   ranks[threadIdx.x] = 0;
   depths[threadIdx.x] = depth;
   sort<edge*edge>(depths, ranks, orders, Far, *maxidx);
   uint32_t mindepth = depths[orders[0]];
   uint32_t maxdepth = depths[orders[*maxidx]];
   const bool haveFar = depths[orders[edge*edge-1]]==Far;
   const uint32_t range = maxdepth-mindepth;
   const float qscale = haveFar ? mask-0.5f : mask+0.5f;

   uint32_t quant = 0;
   if (scalebits==0) {
      if (depth == Far) {
         quant = mask;
      } else if (range > 0) {
         quant = ((depth-mindepth)*qscale)/range;
      }
   } else {
      uint32_t *mididx = &maxidx[blockDim.y];
      const uint32_t scalemask = (1U<<scalebits)-1U;
      const uint32_t depthmask = ~scalemask;
      mindepth &= depthmask;
      maxdepth |= scalemask;
      const uint32_t range = maxdepth-mindepth;
      const uint32_t midval = (mindepth+maxdepth) >> 1;
      findgap<edge*edge>(depths, orders, *maxidx, midval, *mididx);
      const uint32_t lowermid = depths[orders[*mididx]];
      const uint32_t uppermid = depths[orders[(*mididx)+1]];
      int lowerscale=1, upperscale=1;
      if (range > 1) {
         if (lowermid > mindepth)
            lowerscale = range/(lowermid-mindepth)/2;
         if (maxdepth > uppermid)
            upperscale = range/(maxdepth-uppermid)/2;
         if (lowerscale == 0)
            lowerscale = 1;
         if (upperscale == 0)
            upperscale = 1;
         if (lowerscale > 1<<scalebits)
            lowerscale = 1<<scalebits;
         if (upperscale > 1<<scalebits)
            upperscale = 1<<scalebits;
      }

      quant = 0;
      if (depth == Far) {
         quant = mask;
      } else if (range > 0) {
         if (depth <= midval) {
            quant = ((depth-mindepth)*(lowerscale*mask+0.5f))/range;
         } else if (haveFar) {
            quant = ((maxdepth-depth)*(upperscale*(mask-1.5f)))/range;
            quant = mask-1-quant;
         } else {
            quant = ((maxdepth-depth)*(upperscale*mask+0.5f))/range;
            quant = mask-quant;
         }
      }

      --lowerscale;
      --upperscale;

      mindepth &= depthmask;
      mindepth |= lowerscale;
      maxdepth &= depthmask;
      maxdepth |= upperscale;
   }
   depths[threadIdx.x] = quant;

   Quant &sq = ((Quant*)outimg)[(h-1-y)/edge*w/edge+x/edge];
   if (threadIdx.x == 0) {
      if (haveFar) {
         setdepth(sq, 0, maxdepth);
         setdepth(sq, 1, mindepth);
      } else {
         setdepth(sq, 0, mindepth);
         setdepth(sq, 1, maxdepth);
      }

      dq_clearbits(sq);
      for (int iy=0; iy<edge; ++iy) {
         for (int ix=0; ix<edge; ++ix) {
            int i = iy*edge+ix;
            dq_setbits(sq, i*quantbits, quantbits, depths[(edge-1-iy)*edge+ix]);
         }
      }
   }
}


//! retrieve image in YUV format, chrominance planes are optionally sub-sampled
bool ReadBackCuda::readpixelsyuv(GLint x, GLint y, GLint w, GLint pitch, GLint h,
      GLenum format, int ps, GLubyte *bits, GLint buf, int subx, int suby)
{
   size_t subsize = w*h+2*((w+subx-1)/subx)*((h+suby-1)/suby);
   if(!initPbo(pitch*h*4, subsize))
      return false;

   uchar *img = NULL;
#ifdef USE_BGRA
   if (!pbo2cuda(x, y, w, pitch, h, GL_BGRA, ps, buf, GL_UNSIGNED_INT_8_8_8_8_REV, &img))
      return false;
#else
   if (!pbo2cuda(x, y, w, pitch, h, GL_BGR, ps, buf, GL_UNSIGNED_BYTE, &img))
      return false;
#endif

   dim3 grid(((w+TileSizeX-1)/TileSizeX+subx-1)/subx, ((h+TileSizeY-1)/TileSizeY+suby-1)/suby);
   dim3 block(TileSizeX, TileSizeY);

   //fprintf(stderr, "rgb2yuv: x=%d, y=%d, w=%d, h=%d, sub=(%d %d)\n", x, y, w, h, subx, suby);

   rgb2yuv<<<grid, block>>>(img, outImg, w, h, subx, suby);

   cucheck("memcpy", hipMemcpy(bits, outImg, subsize, hipMemcpyDeviceToHost));
   cucheck("unmap res", hipGraphicsUnmapResources(1, &imgRes, NULL));

   //_prof_rb.endframe(w*h, 0, stereo? 0.5 : 1);
   //checkgl("Read Pixels");

   return true;
}

//! RGB read-back
bool ReadBackCuda::readpixels(GLint x, GLint y, GLint w, GLint pitch, GLint h,
      GLenum format, int ps, GLubyte *bits, GLint buf, GLenum type)
{
   //fprintf(stderr, "readpixels: w=%d, pitch=%d\n", w, pitch);
   if(!initPbo(pitch*h*ps, 0))
      return false;

   uchar *img = NULL;
   if (!pbo2cuda(x, y, w, pitch, h, format, ps, buf, type, &img))
      return false;

   if (pitch != w) {
      cucheck("memcpy2d", hipMemcpy2D(bits, pitch*ps, img, pitch*ps, w*ps, h, hipMemcpyDeviceToHost));
   } else {
      cucheck("memcpy", hipMemcpy(bits, img, pitch*h*ps, hipMemcpyDeviceToHost));
   }
   cucheck("unmap res", hipGraphicsUnmapResources(1, &imgRes, NULL));

   //_prof_rb.endframe(w*h, 0, stereo? 0.5 : 1);
   //checkgl("Read Pixels");

   return true;
}

bool ReadBackCuda::pbo2cuda(GLint x, GLint y, GLint w, GLint pitch, GLint h,
      GLenum format, int ps, GLint buf, GLenum type, uchar **img)
{
   if (s_error)
      return false;

   GLint readbuf=GL_BACK;
   glGetIntegerv(GL_READ_BUFFER, &readbuf);

   glReadBuffer(buf);
   glPushClientAttrib(GL_CLIENT_PIXEL_STORE_BIT);

   if(pitch%8==0) glPixelStorei(GL_PACK_ALIGNMENT, 8);
   else if(pitch%4==0) glPixelStorei(GL_PACK_ALIGNMENT, 4);
   else if(pitch%2==0) glPixelStorei(GL_PACK_ALIGNMENT, 2);
   else if(pitch%1==0) glPixelStorei(GL_PACK_ALIGNMENT, 1);

   glPixelStorei(GL_PACK_ROW_LENGTH, pitch);

   int e=glGetError();
   while(e!=GL_NO_ERROR) e=glGetError();  // Clear previous error
   //_prof_rb.startframe();

   glBindBuffer(GL_PIXEL_PACK_BUFFER, pboName);
   glBufferData(GL_PIXEL_PACK_BUFFER, pitch*h*ps, NULL, GL_STREAM_COPY);
   glReadPixels(x, y, w, h, format, type, NULL);
   glBindBuffer(GL_PIXEL_PACK_BUFFER, 0);

   glPopClientAttrib();
   glReadBuffer(readbuf);

   if(!cucheck("map res", hipGraphicsMapResources(1, &imgRes, NULL)))
      return false;

   size_t sz;
   return cucheck("get map ptr", hipGraphicsResourceGetMappedPointer((void **)img, &sz, imgRes));
}

//! compressed depth read-back
bool ReadBackCuda::readdepthquant(GLint x, GLint y, GLint w, GLint pitch, GLint h,
      GLenum format, int ps, GLubyte *bits, GLint buf, GLenum type)
{
   size_t compressed = 0;
   int edge = 1;
   if (ps <= 2) {
      edge = DepthQuantize16::edge;
      compressed = (w+edge-1)/edge*(h+edge-1)/edge;
      compressed *= sizeof(DepthQuantize16);
   } else {
      edge = DepthQuantize24::edge;
      compressed = (w+edge-1)/edge*(h+edge-1)/edge;
      compressed *= sizeof(DepthQuantize24);
   }

#if 0
   if (w%edge || h%edge)
      return false;
#endif

   if(!initPbo(pitch*h*ps, compressed))
      return false;

   uchar *img = NULL;
   if (!pbo2cuda(x, y, w, pitch, h, format, ps, buf, type, &img))
      return false;

   dim3 grid((w+TileSizeX-1)/TileSizeX, (h+TileSizeY-1)/TileSizeY);
   dim3 block(TileSizeX, TileSizeY);

   const size_t sharedsize = 3 * sizeof(uint32_t) * TileSizeX * TileSizeY +
      2 * sizeof(uint32_t)*TileSizeX*TileSizeY/(edge*edge);

   if (format == GL_BGRA) {
      assert(ps == 4);
      depthquant<4, DepthQuantize24, true><<<grid, block, sharedsize>>>(img, outImg, w, h);
   } else {
      switch(ps) {
         case 1:
            depthquant<1, DepthQuantize16, false><<<grid, block, sharedsize>>>(img, outImg, w, h);
            break;
         case 2:
            depthquant<2, DepthQuantize16, false><<<grid, block, sharedsize>>>(img, outImg, w, h);
            break;
         case 3:
            depthquant<3, DepthQuantize24, false><<<grid, block, sharedsize>>>(img, outImg, w, h);
            break;
         case 4:
            depthquant<4, DepthQuantize24, false><<<grid, block, sharedsize>>>(img, outImg, w, h);
            break;
         default:
            fprintf(stderr, "pixel depth not supported\n");
            return false;
      }
   }

   cucheck("depth memcpy", hipMemcpy(bits, outImg, compressed, hipMemcpyDeviceToHost));
   cucheck("depth unmap res", hipGraphicsUnmapResources(1, &imgRes, NULL));

   return true;
}
