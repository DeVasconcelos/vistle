#include "hip/hip_runtime.h"
﻿//
//This code is used for both IsoCut and IsoSurface!
//

#include <sstream>
#include <iomanip>
#include <core/index.h>
#include <core/scalar.h>
#include <core/unstr.h>
#include <core/triangles.h>
#include <core/shm.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include "tables.h"
#include "Leveller.h"

using namespace vistle;

template<typename S>
inline S lerp(S a, S b, Scalar t) {
    return a+t*(b-a);
}

template<Index>
inline Index lerp(Index a, Index b, Scalar t) {
    return t > 0.5 ? b : a;
}


const Scalar EPSILON = 1.0e-10f;
const int MaxNumData = 6;

inline Scalar __host__ __device__ tinterp(Scalar iso, const Scalar &f0, const Scalar &f1) {

   const Scalar diff = (f1 - f0);
   const Scalar d0 = iso - f0;
   if (fabs(diff) < EPSILON) {
       const Scalar d1 = f1 - iso;
      return fabs(d0) < fabs(d1) ? 1 : 0;
   }

   return std::min(Scalar(1), std::max(Scalar(0), d0 / diff));
}

#ifdef CUTTINGSURFACE

//! generate data on the fly for creating cutting surfaces as isosurfaces
struct IsoDataFunctor {

   IsoDataFunctor(const Vector &vertex, const Vector &point, const Vector &direction, const Scalar* x, const Scalar* y, const Scalar* z, int option)
      : m_x(x)
      , m_y(y)
      , m_z(z)
      , m_vertex(vertex)
      , m_point(point)
      , m_direction(direction)
      , m_distance(vertex.dot(point))
      , m_option(option)
      , m_vectorprod(m_direction.cross(m_point-m_vertex))
      , m_cylinderradius2(m_vectorprod.squaredNorm())
      , m_sphereradius2((m_vertex-m_point).squaredNorm())
   {}
   __host__ __device__ Scalar operator()(Index i) {

      switch(m_option) {
         case Plane: {
            Vector coordinates(m_x[i], m_y[i], m_z[i]);
            return m_vertex.dot(coordinates) - m_distance;
         }
         case Sphere: {
            Vector coordinates(m_x[i], m_y[i], m_z[i]);
            return coordinates.squaredNorm() - m_sphereradius2;
         }
         default: {
            // all cylinders
            Vector coordinates(m_x[i], m_y[i], m_z[i]);
            return (m_direction.cross(coordinates - m_vertex)).squaredNorm() - m_cylinderradius2;
         }
      }
   }
   const Scalar* m_x;
   const Scalar* m_y;
   const Scalar* m_z;
   const Vector m_vertex;
   const Vector m_point;
   const Vector m_direction;
   const Scalar m_distance;
   const int m_option;
   const Vector m_vectorprod;
   const Scalar m_cylinderradius2;
   const Scalar m_sphereradius2;
};

#else
//! fetch data from scalar field for generating isosurface
struct IsoDataFunctor {

   IsoDataFunctor(const Scalar *data)
      : m_volumedata(data)
   {}
   __host__ __device__ Scalar operator()(Index i) { return m_volumedata[i]; }

   const Scalar *m_volumedata;

};
#endif

struct HostData {

   Scalar m_isovalue;
   Index m_numinputdata, m_numinputdataI;
   IsoDataFunctor m_isoFunc;
   const Index *m_el;
   const Index *m_cl;
   const unsigned char *m_tl;
   std::vector<Index> m_caseNums;
   std::vector<Index> m_numVertices;
   std::vector<Index> m_LocationList;
   std::vector<Index> m_ValidCellVector;
   const Scalar *m_x;
   const Scalar *m_y;
   const Scalar *m_z;
   std::vector<vistle::shm_ref<vistle::shm_array<Scalar, shm<Scalar>::allocator>>> m_outData;
   std::vector<vistle::shm_ref<vistle::shm_array<Index, shm<Index>::allocator>>> m_outDataI;
   std::vector<const Scalar*> m_inputpointer;
   std::vector<const Index*> m_inputpointerI;
   std::vector<Scalar*> m_outputpointer;
   std::vector<Index *> m_outputpointerI;

   typedef const Index *IndexIterator;
   typedef std::vector<Index>::iterator VectorIndexIterator;

   HostData(Scalar isoValue
            , IsoDataFunctor isoFunc
            , const Index *el
            , const unsigned char *tl
            , const Index *cl
            , const Scalar *x
            , const Scalar *y
            , const Scalar *z
            )
      : m_isovalue(isoValue)
      , m_numinputdata(0)
      , m_numinputdataI(0)
      , m_isoFunc(isoFunc)
      , m_el(el)
      , m_cl(cl)
      , m_tl(tl)
      , m_x(x)
      , m_y(y)
      , m_z(z)
   {
      m_inputpointer.push_back(&x[0]);
      m_inputpointer.push_back(&y[0]);
      m_inputpointer.push_back(&z[0]);

      for(size_t i = 0; i < m_inputpointer.size(); i++){
         m_outData.emplace_back(vistle::ShmVector<Scalar>::create(0));
         m_outputpointer.push_back(NULL);
      }
      m_numinputdata = m_inputpointer.size();
   }

   void addmappeddata(const Scalar *mapdata){

      m_inputpointer.push_back(mapdata);
      m_outData.push_back(vistle::ShmVector<Scalar>::create(0));
      m_outputpointer.push_back(NULL);
      m_numinputdata = m_inputpointer.size();

   }

   void addmappeddata(const Index *mapdata){

      m_inputpointerI.push_back(mapdata);
      m_outDataI.push_back(vistle::ShmVector<Index>::create(0));
      m_outputpointerI.push_back(NULL);
      m_numinputdataI = m_inputpointerI.size();

   }
};

struct DeviceData {

   Scalar m_isovalue;
   Index m_numinputdata, m_numinputdataI;
   IsoDataFunctor m_isoFunc;
   thrust::device_vector<Index> m_el;
   thrust::device_vector<Index> m_cl;
   thrust::device_vector<unsigned char> m_tl;
   thrust::device_vector<Index> m_caseNums;
   thrust::device_vector<Index> m_numVertices;
   thrust::device_vector<Index> m_LocationList;
   thrust::device_vector<Index> m_ValidCellVector;
   thrust::device_vector<Scalar> m_x;
   thrust::device_vector<Scalar> m_y;
   thrust::device_vector<Scalar> m_z;
   std::vector<thrust::device_vector<Scalar> *> m_outData;
   std::vector<thrust::device_vector<Index> *> m_outDataI;
   std::vector<thrust::device_ptr<Scalar> > m_inputpointer;
   std::vector<thrust::device_ptr<Index> > m_inputpointerI;
   std::vector<thrust::device_ptr<Scalar> > m_outputpointer;
   std::vector<thrust::device_ptr<Index> > m_outputpointerI;
   typedef const Index *IndexIterator;
   //typedef thrust::device_vector<Index>::iterator IndexIterator;

   DeviceData(Scalar isoValue
              , IsoDataFunctor isoFunc
              , Index nelem
              , const Index *el
              , const unsigned char *tl
              , Index nconn
              , const Index *cl
              , Index ncoord
              , const Scalar *x
              , const Scalar *y
              , const Scalar *z)
   : m_isovalue(isoValue)
   , m_isoFunc(isoFunc)
   , m_el(el, el+nelem)
   , m_cl(cl, cl+nconn)
   , m_tl(tl, tl+nelem)
   , m_x(x, x+ncoord)
   , m_y(y, y+ncoord)
   , m_z(z, z+ncoord)
   {
      m_inputpointer.push_back(m_x.data());
      m_inputpointer.push_back(m_y.data());
      m_inputpointer.push_back(m_z.data());

      for(size_t i = 0; i < m_inputpointer.size(); i++){
         m_outData.push_back(new thrust::device_vector<Scalar>);
      }
      m_outputpointer.resize(m_inputpointer.size());
      m_numinputdata = m_inputpointer.size();
      for(size_t i = 0; i < m_inputpointerI.size(); i++){
         m_outDataI.push_back(new thrust::device_vector<Index>);
      }
      m_outputpointerI.resize(m_inputpointerI.size());
      m_numinputdataI = m_inputpointerI.size();
   }
};

template<class Data>
struct process_Cell {
   process_Cell(Data &data) : m_data(data) {
      for (int i = 0; i < m_data.m_numinputdata; i++){
         m_data.m_outputpointer[i] = m_data.m_outData[i]->data();
      }
      for (int i = 0; i < m_data.m_numinputdataI; i++){
         m_data.m_outputpointerI[i] = m_data.m_outDataI[i]->data();
      }
   }

   Data &m_data;

   __host__ __device__
   void operator()(Index ValidCellIndex) {

      const Index CellNr = m_data.m_ValidCellVector[ValidCellIndex];
      const Index Cellbegin = m_data.m_el[CellNr];
      const Index Cellend = m_data.m_el[CellNr+1];
      const Index numVert = m_data.m_numVertices[ValidCellIndex];
      const auto &cl = &m_data.m_cl[Cellbegin];

#define INTER(triTable, edgeTable) \
    const unsigned int edge = triTable[m_data.m_caseNums[ValidCellIndex]][idx]; \
    const unsigned int v1 = edgeTable[0][edge]; \
    const unsigned int v2 = edgeTable[1][edge]; \
    const Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]); \
    Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx; \
    for(Index j = 0; j < m_data.m_numinputdata; j++) { \
        m_data.m_outputpointer[j][outvertexindex] = \
            lerp(m_data.m_inputpointer[j][cl[v1]], m_data.m_inputpointer[j][cl[v2]], t); \
    } \
    for(Index j = 0; j < m_data.m_numinputdataI; j++) { \
        m_data.m_outputpointerI[j][outvertexindex] = \
            lerp(m_data.m_inputpointerI[j][cl[v1]], m_data.m_inputpointerI[j][cl[v2]], t); \
    }

      switch (m_data.m_tl[CellNr]) {

         case UnstructuredGrid::HEXAHEDRON: {

            Scalar field[8];
            for (int idx = 0; idx < 8; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
               INTER(hexaTriTable, hexaEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::TETRAHEDRON: {

            Scalar field[4];
            for (int idx = 0; idx < 4; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
               INTER(tetraTriTable, tetraEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::PYRAMID: {

            Scalar field[5];
            for (int idx = 0; idx < 5; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
                INTER(pyrTriTable, pyrEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::PRISM: {

            Scalar field[6];
            for (int idx = 0; idx < 6; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
                INTER(prismTriTable, prismEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::POLYHEDRON: {

            Index sidebegin = InvalidIndex;
            bool vertexSaved = false;
            Scalar savedData [MaxNumData];
            Index savedDataI[MaxNumData];
            Index numTri = 0;
            int flag = 0;
            Scalar middleData[MaxNumData];
            Index middleDataI[MaxNumData];
            for(int i = 0; i < MaxNumData; i++ ){
               middleData[i] = 0;
               middleDataI[i] = 0;
            };
            Scalar cd1[MaxNumData], cd2[MaxNumData];
            Index cd1I[MaxNumData], cd2I[MaxNumData];

            Index outIdx = m_data.m_LocationList[ValidCellIndex];
            for (Index i = Cellbegin; i < Cellend; i++) {

               const Index c1 = m_data.m_cl[i];
               const Index c2 = m_data.m_cl[i+1];

               if (c1 == sidebegin) {

                  sidebegin = InvalidIndex;
                  if (vertexSaved) {

                     for(Index i = 0; i < m_data.m_numinputdata; i++){
                        m_data.m_outputpointer[i][outIdx] = savedData[i];
                     }
                     for(Index i = 0; i < m_data.m_numinputdataI; i++){
                        m_data.m_outputpointerI[i][outIdx] = savedDataI[i];
                     }

                     outIdx += 2;
                     vertexSaved=false;
                  }
                  continue;
               } else if(sidebegin == InvalidIndex) { //Wenn die Neue Seite beginnt

                  flag = 0;
                  sidebegin = c1;
                  vertexSaved = false;
               }

               for(int i = 0; i < m_data.m_numinputdata; i++){
                  cd1[i] = m_data.m_inputpointer[i][c1];
                  cd2[i] = m_data.m_inputpointer[i][c2];
               }
               for(int i = 0; i < m_data.m_numinputdataI; i++){
                  cd1I[i] = m_data.m_inputpointerI[i][c1];
                  cd2I[i] = m_data.m_inputpointerI[i][c2];
               }

               Scalar d1 = m_data.m_isoFunc(c1);
               Scalar d2 = m_data.m_isoFunc(c2);
               Scalar t = tinterp(m_data.m_isovalue, d1, d2);

               if (d1 <= m_data.m_isovalue && d2 > m_data.m_isovalue) {
                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     Scalar v = lerp(cd1[i], cd2[i], t);
                     middleData[i] += v;
                     m_data.m_outputpointer[i][outIdx] = v;
                  }
                  for(Index i = 0; i < m_data.m_numinputdataI; i++){
                     Index vI = lerp(cd1I[i], cd2I[i], t);
                     middleDataI[i] += vI;
                     m_data.m_outputpointerI[i][outIdx] = vI;
                  };

                  ++outIdx;
                  ++numTri;
                  flag = 1;

               } else if (d1 > m_data.m_isovalue && d2 <= m_data.m_isovalue) {

                  Scalar v [MaxNumData];
                  Index vI[MaxNumData];

                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     v[i] = lerp(cd1[i], cd2[i], t);
                     middleData[i] += v[i];
                  }
                  for(Index i = 0; i < m_data.m_numinputdataI; i++){
                     vI[i] = lerp(cd1I[i], cd2I[i], t);
                     middleDataI[i] += vI[i];
                  }
                  ++numTri;
                  if (flag == 1) { //fall 2 nach fall 1
                     for(Index i = 0; i < m_data.m_numinputdata; i++){
                        m_data.m_outputpointer[i][outIdx] = v[i];
                     }
                     for(Index i = 0; i < m_data.m_numinputdataI; i++){
                        m_data.m_outputpointerI[i][outIdx] = vI[i];
                     }
                     outIdx += 2;
                  } else { //fall 2 zuerst

                     for(Index i = 0; i < m_data.m_numinputdata; i++){
                        savedData[i] = v[i];
                     }
                     for(Index i = 0; i < m_data.m_numinputdataI; i++){
                        savedDataI[i] = vI[i];
                     }
                     vertexSaved=true;
                  }
               }
            }
            if (numTri > 0) {
                for(Index i = 0; i < m_data.m_numinputdata; i++){
                    middleData[i] /= numTri;
                }
                for(Index i = 0; i < m_data.m_numinputdataI; i++){
                    middleDataI[i] /= numTri;
                }
            }
            for (Index i = 2; i < numVert; i += 3) {
               const Index idx = m_data.m_LocationList[ValidCellIndex]+i;
               for(Index i = 0; i < m_data.m_numinputdata; i++){
                  m_data.m_outputpointer[i][idx] = middleData[i];
               }
               for(Index i = 0; i < m_data.m_numinputdataI; i++){
                  m_data.m_outputpointerI[i][idx] = middleDataI[i];
               }
            };
            break;
         }
      }
   }
};

template<class Data>
struct checkcell {

   typedef float argument_type;
   typedef float result_type;
   Data &m_data;
   checkcell(Data &data) : m_data(data) {}

   __host__ __device__ int operator()(const thrust::tuple<Index,Index> iCell) const {

      int havelower = 0;
      int havehigher = 0;
      Index Cell = iCell.get<0>();
      Index nextCell = iCell.get<1>();
      for (Index i=Cell; i<nextCell; i++) {
         float val = m_data.m_isoFunc(m_data.m_cl[i]);
         if (val>m_data.m_isovalue) {
            havelower=1;
            if (havehigher)
               return 1;
         } else {
            havehigher=1;
            if (havelower)
               return 1;
         }
      }
      return 0;
   }
};

template<class Data>
struct classify_cell {

   classify_cell(Data &data) : m_data(data) {}

   Data &m_data;

   __host__ __device__ thrust::tuple<Index,Index> operator()(Index CellNr) {

      uint tableIndex = 0;
      Index Start = m_data.m_el[CellNr];
      Index diff = m_data.m_el[CellNr+1]-Start;
      unsigned char CellType = m_data.m_tl[CellNr];
      int numVerts = 0;
      if (CellType != UnstructuredGrid::POLYHEDRON) {
         for (int idx = 0; idx < diff; idx ++) {
            tableIndex += (((int) (m_data.m_isoFunc(m_data.m_cl[Start+idx]) > m_data.m_isovalue)) << idx);
         }
      }
      switch (CellType) {

         case UnstructuredGrid::HEXAHEDRON:
            numVerts = hexaNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::TETRAHEDRON:
            numVerts = tetraNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PYRAMID:
            numVerts = pyrNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PRISM:
            numVerts = prismNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::POLYHEDRON: {

            Index sidebegin = InvalidIndex;
            Index vertcounter = 0;
            for (Index i = Start; i < Start + diff; i++) {

               if (m_data.m_cl[i] == sidebegin) {
                  sidebegin = InvalidIndex;
                  continue;
               }

               if (sidebegin == InvalidIndex) {
                  sidebegin = m_data.m_cl[i];
               }

               if (m_data.m_isoFunc(m_data.m_cl[i]) <= m_data.m_isovalue && m_data.m_isoFunc(m_data.m_cl[i+1]) > m_data.m_isovalue) {

                  vertcounter += 1;
               } else if(m_data.m_isoFunc(m_data.m_cl[i]) > m_data.m_isovalue && m_data.m_isoFunc(m_data.m_cl[i+1]) <= m_data.m_isovalue) {

                  vertcounter += 1;
               }
            }
            numVerts = vertcounter + vertcounter/2;
            break;
         }
      };
      return thrust::make_tuple<Index, Index> (tableIndex, numVerts);
   }
};

Leveller::Leveller(UnstructuredGrid::const_ptr grid, const Scalar isovalue, Index processortype
         #ifdef CUTTINGSURFACE
            , int option
         #endif
            )
      : m_grid(grid)
   #ifdef CUTTINGSURFACE
      , m_option(option)
   #endif
      , m_isoValue(isovalue)
      , m_processortype(processortype)
      , gmin(std::numeric_limits<Scalar>::max())
      , gmax(-std::numeric_limits<Scalar>::max())
   {
      m_triangles = Triangles::ptr(new Triangles(Object::Initialized));
      m_triangles->setMeta(grid->meta());
   }

template<class Data, class pol>
Index Leveller::calculateSurface(Data &data) {

   thrust::counting_iterator<int> first(0);
   thrust::counting_iterator<int> last = first + m_grid->getNumElements();
   typedef thrust::tuple<typename Data::IndexIterator, typename Data::IndexIterator> Iteratortuple;
   typedef thrust::zip_iterator<Iteratortuple> ZipIterator;
   ZipIterator ElTupleVec(thrust::make_tuple(&data.m_el[0], &data.m_el[1]));
   data.m_ValidCellVector.resize(m_grid->getNumElements());
   typename Data::VectorIndexIterator end = thrust::copy_if(pol(), first, last, ElTupleVec, data.m_ValidCellVector.begin(), checkcell<Data>(data));
   size_t numValidCells = end-data.m_ValidCellVector.begin();
   data.m_caseNums.resize(numValidCells);
   data.m_numVertices.resize(numValidCells);
   data.m_LocationList.resize(numValidCells);
   thrust::transform(pol(), data.m_ValidCellVector.begin(), end, thrust::make_zip_iterator(thrust::make_tuple(data.m_caseNums.begin(),data.m_numVertices.begin())), classify_cell<Data>(data));
   thrust::exclusive_scan(pol(), data.m_numVertices.begin(), data.m_numVertices.end(), data.m_LocationList.begin());
   Index totalNumVertices = 0;
   if (!data.m_numVertices.empty())
      totalNumVertices += data.m_numVertices.back();
   if (!data.m_LocationList.empty())
      totalNumVertices += data.m_LocationList.back();
   for(int i = 0; i < data.m_numinputdata; i++){
      data.m_outData[i]->resize(totalNumVertices);
   }
   for(int i = 0; i < data.m_numinputdataI; i++){
      data.m_outDataI[i]->resize(totalNumVertices);
   }
   thrust::counting_iterator<Index> start(0), finish(numValidCells);
   thrust::for_each(pol(), start, finish, process_Cell<Data>(data));
   return totalNumVertices;
}

bool Leveller::process() {
   if(m_mapdata.size()){
      Vec<Scalar>::const_ptr mapdataobj = Vec<Scalar>::as(m_mapdata[0]);
   }
#ifndef CUTTINGSURFACE
   Vec<Scalar>::const_ptr dataobj = Vec<Scalar>::as(m_data);
   if (!dataobj)
      return false;
#else
#endif

   Index totalNumVertices = 0;

   switch (m_processortype) {

      case Host: {

         HostData HD(m_isoValue,
#ifndef CUTTINGSURFACE
               IsoDataFunctor(&dataobj->x()[0]),
#else
               IsoDataFunctor(vertex, point, direction, &m_grid->x()[0], &m_grid->y()[0], &m_grid->z()[0], m_option),
#endif
               m_grid->el(), m_grid->tl(), m_grid->cl(), m_grid->x(), m_grid->y(), m_grid->z());

         if(m_mapdata.size()){
            if(Vec<Scalar,1>::const_ptr Scal = Vec<Scalar,1>::as(m_mapdata[0])){
               HD.addmappeddata(Scal->x());
            }
            if(Vec<Scalar,3>::const_ptr Vect = Vec<Scalar,3>::as(m_mapdata[0])){
               HD.addmappeddata(Vect->x());
               HD.addmappeddata(Vect->y());
               HD.addmappeddata(Vect->z());
            }
            if(Vec<Index,1>::const_ptr Idx = Vec<Index,1>::as(m_mapdata[0])){
               HD.addmappeddata(Idx->x());
            }

         }

         totalNumVertices = calculateSurface<HostData, thrust::detail::host_t>(HD);

         m_triangles->d()->x[0] = HD.m_outData[0];
         m_triangles->d()->x[1] = HD.m_outData[1];
         m_triangles->d()->x[2] = HD.m_outData[2];

         if(m_mapdata.size()){
            if(Vec<Scalar>::as(m_mapdata[0])){

               Vec<Scalar,1>::ptr out = Vec<Scalar,1>::ptr(new Vec<Scalar,1>(Object::Initialized));
               out->d()->x[0] = HD.m_outData[3];
               out->setMeta(m_mapdata[0]->meta());
               m_outmapData.push_back(out);

            }
            if(Vec<Scalar,3>::as(m_mapdata[0])){

               Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(Object::Initialized));
               out->d()->x[0] = HD.m_outData[3];
               out->d()->x[1] = HD.m_outData[4];
               out->d()->x[2] = HD.m_outData[5];
               out->setMeta(m_mapdata[0]->meta());
               m_outmapData.push_back(out);

            }
            if(Vec<Index>::as(m_mapdata[0])){

               Vec<Index>::ptr out = Vec<Index>::ptr(new Vec<Index>(Object::Initialized));
               out->d()->x[0] = HD.m_outDataI[0];
               out->setMeta(m_mapdata[0]->meta());
               m_outmapData.push_back(out);

            }
         }
         break;
      }

      case Device: {

         DeviceData DD(m_isoValue,
#ifndef CUTTINGSURFACE
               IsoDataFunctor(&dataobj->x()[0]),
#else
               IsoDataFunctor(vertex, point, direction, &m_grid->x()[0], &m_grid->y()[0], &m_grid->z()[0], m_option),
#endif
               m_grid->getNumElements(), m_grid->el(), m_grid->tl(), m_grid->getNumCorners(), m_grid->cl(), m_grid->getSize(), m_grid->x(), m_grid->y(), m_grid->z());

#if 0
         totalNumVertices = calculateSurface<DeviceData, thrust::device>(DD);
#endif

         m_triangles->x().resize(totalNumVertices);
         Scalar *out_x = m_triangles->x().data();
         thrust::copy(DD.m_outData[0]->begin(), DD.m_outData[0]->end(), out_x);

         m_triangles->y().resize(totalNumVertices);
         Scalar *out_y = m_triangles->y().data();
         thrust::copy(DD.m_outData[1]->begin(), DD.m_outData[1]->end(), out_y);

         m_triangles->z().resize(totalNumVertices);
         Scalar *out_z = m_triangles->z().data();
         thrust::copy(DD.m_outData[2]->begin(), DD.m_outData[2]->end(), out_z);

         if(m_mapdata.size()){
            if(Vec<Scalar>::as(m_mapdata[0])){

               Vec<Scalar>::ptr out = Vec<Scalar>::ptr(new Vec<Scalar>(totalNumVertices));
               thrust::copy(DD.m_outData[3]->begin(), DD.m_outData[3]->end(), out->x().data());
               out->setMeta(m_mapdata[0]->meta());
               m_outmapData.push_back(out);

            }
            if(Vec<Scalar,3>::as(m_mapdata[0])){

               Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(totalNumVertices));
               thrust::copy(DD.m_outData[3]->begin(), DD.m_outData[3]->end(), out->x().data());
               thrust::copy(DD.m_outData[4]->begin(), DD.m_outData[4]->end(), out->y().data());
               thrust::copy(DD.m_outData[5]->begin(), DD.m_outData[5]->end(), out->z().data());
               out->setMeta(m_mapdata[0]->meta());
               m_outmapData.push_back(out);

            }
         }
         break;
      }
   }

   return true;
}

#ifdef CUTTINGSURFACE
void Leveller::setCutData(const Vector m_vertex, const Vector m_point, const Vector m_direction){
   vertex = m_vertex;
   point = m_point;
   direction = m_direction;
}
#else    
void Leveller::setIsoData(Vec<Scalar>::const_ptr obj) {
   m_data = obj;
}
#endif

void Leveller::addMappedData(Object::const_ptr mapobj ){
   m_mapdata.push_back(mapobj);
}

Object::ptr Leveller::result() {
      return m_triangles;
   }

DataBase::ptr Leveller::mapresult() {
   if(m_outmapData.size())
      return m_outmapData[0];
   else
      return DataBase::ptr();
}

std::pair<Scalar, Scalar> Leveller::range() {
   return std::make_pair(gmin, gmax);
}
