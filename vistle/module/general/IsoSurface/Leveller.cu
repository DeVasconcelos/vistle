#include "hip/hip_runtime.h"
﻿//
//This code is used for both IsoCut and IsoSurface!
//

#include <sstream>
#include <iomanip>
#include <core/index.h>
#include <core/scalar.h>
#include <core/unstr.h>
#include <core/triangles.h>
#include <core/shm.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include "tables.h"
#include "Leveller.h"

using namespace vistle;

template<typename S>
inline S lerp(S a, S b, Scalar t) {
    return a+t*(b-a);
}

template<Index>
inline Index lerp(Index a, Index b, Scalar t) {
    return t > 0.5 ? b : a;
}


const Scalar EPSILON = 1.0e-10f;
const int MaxNumData = 6;

inline Scalar __host__ __device__ tinterp(Scalar iso, const Scalar &f0, const Scalar &f1) {

   const Scalar diff = (f1 - f0);
   const Scalar d0 = iso - f0;
   if (fabs(diff) < EPSILON) {
       const Scalar d1 = f1 - iso;
      return fabs(d0) < fabs(d1) ? 1 : 0;
   }

   return std::min(Scalar(1), std::max(Scalar(0), d0 / diff));
}

#ifdef CUTTINGSURFACE

//! generate data on the fly for creating cutting surfaces as isosurfaces
struct IsoDataFunctor {

   IsoDataFunctor(const Vector &vertex, const Vector &point, const Vector &direction, const Scalar* x, const Scalar* y, const Scalar* z, int option)
      : m_x(x)
      , m_y(y)
      , m_z(z)
      , m_vertex(vertex)
      , m_point(point)
      , m_direction(direction)
      , m_distance(vertex.dot(point))
      , m_option(option)
      , m_radius2((m_option==Sphere ? m_point-m_vertex : m_direction.cross(m_point-m_vertex)).squaredNorm())
   {}

   __host__ __device__ Scalar operator()(Index i) {
      Vector coordinates(m_x[i], m_y[i], m_z[i]);
      switch(m_option) {
         case Plane: {
            return m_vertex.dot(coordinates) - m_distance;
         }
         case Sphere: {
            return (coordinates-m_vertex).squaredNorm() - m_radius2;
         }
         default: {
            // all cylinders
            return (m_direction.cross(coordinates - m_vertex)).squaredNorm() - m_radius2;
         }
      }
   }
   const Scalar* m_x;
   const Scalar* m_y;
   const Scalar* m_z;
   const Vector m_vertex;
   const Vector m_point;
   const Vector m_direction;
   const Scalar m_distance;
   const int m_option;
   const Scalar m_radius2;
};

#else
//! fetch data from scalar field for generating isosurface
struct IsoDataFunctor {

   IsoDataFunctor(const Scalar *data)
      : m_volumedata(data)
   {}
   __host__ __device__ Scalar operator()(Index i) { return m_volumedata[i]; }

   const Scalar *m_volumedata;

};
#endif

struct HostData {

   Scalar m_isovalue;
   Index m_numInVertData, m_numInVertDataI;
   Index m_numInCellData, m_numInCellDataI;
   IsoDataFunctor m_isoFunc;
   const Index *m_el;
   const Index *m_cl;
   const unsigned char *m_tl;
   std::vector<Index> m_caseNums;
   std::vector<Index> m_numVertices;
   std::vector<Index> m_LocationList;
   std::vector<Index> m_ValidCellVector;
   const Scalar *m_x;
   const Scalar *m_y;
   const Scalar *m_z;
   std::vector<vistle::shm_ref<vistle::shm_array<Scalar, shm<Scalar>::allocator>>> m_outVertData, m_outCellData;
   std::vector<vistle::shm_ref<vistle::shm_array<Index, shm<Index>::allocator>>> m_outVertDataI, m_outCellDataI;
   std::vector<const Scalar*> m_inVertPtr, m_inCellPtr;
   std::vector<const Index*> m_inVertPtrI, m_inCellPtrI;
   std::vector<Scalar*> m_outVertPtr, m_outCellPtr;
   std::vector<Index *> m_outVertPtrI, m_outCellPtrI;

   typedef const Index *IndexIterator;
   typedef std::vector<Index>::iterator VectorIndexIterator;

   HostData(Scalar isoValue
            , IsoDataFunctor isoFunc
            , const Index *el
            , const unsigned char *tl
            , const Index *cl
            , const Scalar *x
            , const Scalar *y
            , const Scalar *z
            )
      : m_isovalue(isoValue)
      , m_numInVertData(0)
      , m_numInVertDataI(0)
      , m_numInCellData(0)
      , m_numInCellDataI(0)
      , m_isoFunc(isoFunc)
      , m_el(el)
      , m_cl(cl)
      , m_tl(tl)
      , m_x(x)
      , m_y(y)
      , m_z(z)
   {
      m_inVertPtr.push_back(&x[0]);
      m_inVertPtr.push_back(&y[0]);
      m_inVertPtr.push_back(&z[0]);

      for(size_t i = 0; i < m_inVertPtr.size(); i++){
         m_outVertData.emplace_back(vistle::ShmVector<Scalar>::create(0));
         m_outVertPtr.push_back(NULL);
      }
      m_numInVertData = m_inVertPtr.size();
   }

   void addmappeddata(const Scalar *mapdata){

      m_inVertPtr.push_back(mapdata);
      m_outVertData.push_back(vistle::ShmVector<Scalar>::create(0));
      m_outVertPtr.push_back(NULL);
      m_numInVertData = m_inVertPtr.size();
   }

   void addmappeddata(const Index *mapdata){

      m_inVertPtrI.push_back(mapdata);
      m_outVertDataI.push_back(vistle::ShmVector<Index>::create(0));
      m_outVertPtrI.push_back(NULL);
      m_numInVertDataI = m_inVertPtrI.size();
   }

   void addcelldata(const Scalar *mapdata){

      m_inCellPtr.push_back(mapdata);
      m_outCellData.push_back(vistle::ShmVector<Scalar>::create(0));
      m_outCellPtr.push_back(NULL);
      m_numInCellData = m_inCellPtr.size();
   }

   void addcelldata(const Index *mapdata){

      m_inCellPtrI.push_back(mapdata);
      m_outCellDataI.push_back(vistle::ShmVector<Index>::create(0));
      m_outCellPtrI.push_back(NULL);
      m_numInCellDataI = m_inCellPtrI.size();
   }
};

struct DeviceData {

   Scalar m_isovalue;
   Index m_numInVertData, m_numInVertDataI;
   Index m_numInCellData, m_numInCellDataI;
   IsoDataFunctor m_isoFunc;
   thrust::device_vector<Index> m_el;
   thrust::device_vector<Index> m_cl;
   thrust::device_vector<unsigned char> m_tl;
   thrust::device_vector<Index> m_caseNums;
   thrust::device_vector<Index> m_numVertices;
   thrust::device_vector<Index> m_LocationList;
   thrust::device_vector<Index> m_ValidCellVector;
   thrust::device_vector<Scalar> m_x;
   thrust::device_vector<Scalar> m_y;
   thrust::device_vector<Scalar> m_z;
   std::vector<thrust::device_vector<Scalar> *> m_outVertData, m_outCellData;
   std::vector<thrust::device_vector<Index> *> m_outVertDataI, m_outCellDataI;
   std::vector<thrust::device_ptr<Scalar> > m_inVertPtr, m_inCellPtr;
   std::vector<thrust::device_ptr<Index> > m_inVertPtrI, m_inCellPtrI;
   std::vector<thrust::device_ptr<Scalar> > m_outVertPtr, m_outCellPtr;
   std::vector<thrust::device_ptr<Index> > m_outVertPtrI, m_outCellPtrI;
   typedef const Index *IndexIterator;
   //typedef thrust::device_vector<Index>::iterator IndexIterator;

   DeviceData(Scalar isoValue
              , IsoDataFunctor isoFunc
              , Index nelem
              , const Index *el
              , const unsigned char *tl
              , Index nconn
              , const Index *cl
              , Index ncoord
              , const Scalar *x
              , const Scalar *y
              , const Scalar *z)
   : m_isovalue(isoValue)
   , m_isoFunc(isoFunc)
   , m_el(el, el+nelem)
   , m_cl(cl, cl+nconn)
   , m_tl(tl, tl+nelem)
   , m_x(x, x+ncoord)
   , m_y(y, y+ncoord)
   , m_z(z, z+ncoord)
   {
      m_inVertPtr.push_back(m_x.data());
      m_inVertPtr.push_back(m_y.data());
      m_inVertPtr.push_back(m_z.data());

      for(size_t i = 0; i < m_inVertPtr.size(); i++){
         m_outVertData.push_back(new thrust::device_vector<Scalar>);
      }
      m_outVertPtr.resize(m_inVertPtr.size());
      m_numInVertData = m_inVertPtr.size();
      for(size_t i = 0; i < m_inVertPtrI.size(); i++){
         m_outVertDataI.push_back(new thrust::device_vector<Index>);
      }
      m_outVertPtrI.resize(m_inVertPtrI.size());
      m_numInVertDataI = m_inVertPtrI.size();
   }
};

template<class Data>
struct process_Cell {
   process_Cell(Data &data) : m_data(data) {
      for (int i = 0; i < m_data.m_numInVertData; i++){
         m_data.m_outVertPtr[i] = m_data.m_outVertData[i]->data();
      }
      for (int i = 0; i < m_data.m_numInVertDataI; i++){
         m_data.m_outVertPtrI[i] = m_data.m_outVertDataI[i]->data();
      }
      for (int i = 0; i < m_data.m_numInCellData; i++){
         m_data.m_outCellPtr[i] = m_data.m_outCellData[i]->data();
      }
      for (int i = 0; i < m_data.m_numInCellDataI; i++){
         m_data.m_outCellPtrI[i] = m_data.m_outCellDataI[i]->data();
      }
   }

   Data &m_data;

   __host__ __device__
   void operator()(Index ValidCellIndex) {

      const Index CellNr = m_data.m_ValidCellVector[ValidCellIndex];
      const Index Cellbegin = m_data.m_el[CellNr];
      const Index Cellend = m_data.m_el[CellNr+1];
      const Index numVert = m_data.m_numVertices[ValidCellIndex];
      const auto &cl = &m_data.m_cl[Cellbegin];

#define INTER(triTable, edgeTable) \
    const unsigned int edge = triTable[m_data.m_caseNums[ValidCellIndex]][idx]; \
    const unsigned int v1 = edgeTable[0][edge]; \
    const unsigned int v2 = edgeTable[1][edge]; \
    const Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]); \
    Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx; \
    for(Index j = 0; j < m_data.m_numInVertData; j++) { \
        m_data.m_outVertPtr[j][outvertexindex] = \
            lerp(m_data.m_inVertPtr[j][cl[v1]], m_data.m_inVertPtr[j][cl[v2]], t); \
    } \
    for(Index j = 0; j < m_data.m_numInVertDataI; j++) { \
        m_data.m_outVertPtrI[j][outvertexindex] = \
            lerp(m_data.m_inVertPtrI[j][cl[v1]], m_data.m_inVertPtrI[j][cl[v2]], t); \
    }

      for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]/3; idx++) {
          Index outcellindex = m_data.m_LocationList[ValidCellIndex]/3+idx; \
          for(Index j = 0; j < m_data.m_numInCellData; j++) {
              m_data.m_outCellPtr[j][outcellindex] = m_data.m_inCellPtr[j][CellNr];
          }
          for(Index j = 0; j < m_data.m_numInCellDataI; j++) {
              m_data.m_outCellPtrI[j][outcellindex] = m_data.m_inCellPtrI[j][CellNr];
          }
      }

      switch (m_data.m_tl[CellNr]) {

         case UnstructuredGrid::HEXAHEDRON: {

            Scalar field[8];
            for (int idx = 0; idx < 8; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
               INTER(hexaTriTable, hexaEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::TETRAHEDRON: {

            Scalar field[4];
            for (int idx = 0; idx < 4; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
               INTER(tetraTriTable, tetraEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::PYRAMID: {

            Scalar field[5];
            for (int idx = 0; idx < 5; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
                INTER(pyrTriTable, pyrEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::PRISM: {

            Scalar field[6];
            for (int idx = 0; idx < 6; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {
                INTER(prismTriTable, prismEdgeTable);
            }
            break;
         }

         case UnstructuredGrid::POLYHEDRON: {

            Index sidebegin = InvalidIndex;
            bool vertexSaved = false;
            Scalar savedData [MaxNumData];
            Index savedDataI[MaxNumData];
            Index numTri = 0;
            int flag = 0;
            Scalar middleData[MaxNumData];
            Index middleDataI[MaxNumData];
            for(int i = 0; i < MaxNumData; i++ ){
               middleData[i] = 0;
               middleDataI[i] = 0;
            };
            Scalar cd1[MaxNumData], cd2[MaxNumData];
            Index cd1I[MaxNumData], cd2I[MaxNumData];

            Index outIdx = m_data.m_LocationList[ValidCellIndex];
            for (Index i = Cellbegin; i < Cellend; i++) {

               const Index c1 = m_data.m_cl[i];
               const Index c2 = m_data.m_cl[i+1];

               if (c1 == sidebegin) {

                  sidebegin = InvalidIndex;
                  if (vertexSaved) {

                     for(Index i = 0; i < m_data.m_numInVertData; i++){
                        m_data.m_outVertPtr[i][outIdx] = savedData[i];
                     }
                     for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                        m_data.m_outVertPtrI[i][outIdx] = savedDataI[i];
                     }

                     outIdx += 2;
                     vertexSaved=false;
                  }
                  continue;
               } else if(sidebegin == InvalidIndex) { //Wenn die Neue Seite beginnt

                  flag = 0;
                  sidebegin = c1;
                  vertexSaved = false;
               }

               for(int i = 0; i < m_data.m_numInVertData; i++){
                  cd1[i] = m_data.m_inVertPtr[i][c1];
                  cd2[i] = m_data.m_inVertPtr[i][c2];
               }
               for(int i = 0; i < m_data.m_numInVertDataI; i++){
                  cd1I[i] = m_data.m_inVertPtrI[i][c1];
                  cd2I[i] = m_data.m_inVertPtrI[i][c2];
               }

               Scalar d1 = m_data.m_isoFunc(c1);
               Scalar d2 = m_data.m_isoFunc(c2);
               Scalar t = tinterp(m_data.m_isovalue, d1, d2);

               if (d1 <= m_data.m_isovalue && d2 > m_data.m_isovalue) {
                  for(Index i = 0; i < m_data.m_numInVertData; i++){
                     Scalar v = lerp(cd1[i], cd2[i], t);
                     middleData[i] += v;
                     m_data.m_outVertPtr[i][outIdx] = v;
                  }
                  for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                     Index vI = lerp(cd1I[i], cd2I[i], t);
                     middleDataI[i] += vI;
                     m_data.m_outVertPtrI[i][outIdx] = vI;
                  };

                  ++outIdx;
                  ++numTri;
                  flag = 1;

               } else if (d1 > m_data.m_isovalue && d2 <= m_data.m_isovalue) {

                  Scalar v [MaxNumData];
                  Index vI[MaxNumData];

                  for(Index i = 0; i < m_data.m_numInVertData; i++){
                     v[i] = lerp(cd1[i], cd2[i], t);
                     middleData[i] += v[i];
                  }
                  for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                     vI[i] = lerp(cd1I[i], cd2I[i], t);
                     middleDataI[i] += vI[i];
                  }
                  ++numTri;
                  if (flag == 1) { //fall 2 nach fall 1
                     for(Index i = 0; i < m_data.m_numInVertData; i++){
                        m_data.m_outVertPtr[i][outIdx] = v[i];
                     }
                     for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                        m_data.m_outVertPtrI[i][outIdx] = vI[i];
                     }
                     outIdx += 2;
                  } else { //fall 2 zuerst

                     for(Index i = 0; i < m_data.m_numInVertData; i++){
                        savedData[i] = v[i];
                     }
                     for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                        savedDataI[i] = vI[i];
                     }
                     vertexSaved=true;
                  }
               }
            }
            if (numTri > 0) {
                for(Index i = 0; i < m_data.m_numInVertData; i++){
                    middleData[i] /= numTri;
                }
                for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                    middleDataI[i] /= numTri;
                }
            }
            for (Index i = 2; i < numVert; i += 3) {
               const Index idx = m_data.m_LocationList[ValidCellIndex]+i;
               for(Index i = 0; i < m_data.m_numInVertData; i++){
                  m_data.m_outVertPtr[i][idx] = middleData[i];
               }
               for(Index i = 0; i < m_data.m_numInVertDataI; i++){
                  m_data.m_outVertPtrI[i][idx] = middleDataI[i];
               }
            };
            break;
         }
      }
   }
};

template<class Data>
struct checkcell {

   typedef float argument_type;
   typedef float result_type;
   Data &m_data;
   checkcell(Data &data) : m_data(data) {}

   __host__ __device__ int operator()(const thrust::tuple<Index,Index> iCell) const {

      int havelower = 0;
      int havehigher = 0;
      Index Cell = iCell.get<0>();
      Index nextCell = iCell.get<1>();
      for (Index i=Cell; i<nextCell; i++) {
         float val = m_data.m_isoFunc(m_data.m_cl[i]);
         if (val>m_data.m_isovalue) {
            havelower=1;
            if (havehigher)
               return 1;
         } else {
            havehigher=1;
            if (havelower)
               return 1;
         }
      }
      return 0;
   }
};

template<class Data>
struct classify_cell {

   classify_cell(Data &data) : m_data(data) {}

   Data &m_data;

   __host__ __device__ thrust::tuple<Index,Index> operator()(Index CellNr) {

      uint tableIndex = 0;
      Index Start = m_data.m_el[CellNr];
      Index diff = m_data.m_el[CellNr+1]-Start;
      unsigned char CellType = m_data.m_tl[CellNr];
      int numVerts = 0;
      if (CellType != UnstructuredGrid::POLYHEDRON) {
         for (int idx = 0; idx < diff; idx ++) {
            tableIndex += (((int) (m_data.m_isoFunc(m_data.m_cl[Start+idx]) > m_data.m_isovalue)) << idx);
         }
      }
      switch (CellType) {

         case UnstructuredGrid::HEXAHEDRON:
            numVerts = hexaNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::TETRAHEDRON:
            numVerts = tetraNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PYRAMID:
            numVerts = pyrNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PRISM:
            numVerts = prismNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::POLYHEDRON: {

            Index sidebegin = InvalidIndex;
            Index vertcounter = 0;
            for (Index i = Start; i < Start + diff; i++) {

               if (m_data.m_cl[i] == sidebegin) {
                  sidebegin = InvalidIndex;
                  continue;
               }

               if (sidebegin == InvalidIndex) {
                  sidebegin = m_data.m_cl[i];
               }

               if (m_data.m_isoFunc(m_data.m_cl[i]) <= m_data.m_isovalue && m_data.m_isoFunc(m_data.m_cl[i+1]) > m_data.m_isovalue) {

                  vertcounter += 1;
               } else if(m_data.m_isoFunc(m_data.m_cl[i]) > m_data.m_isovalue && m_data.m_isoFunc(m_data.m_cl[i+1]) <= m_data.m_isovalue) {

                  vertcounter += 1;
               }
            }
            numVerts = vertcounter + vertcounter/2;
            break;
         }
      };
      return thrust::make_tuple<Index, Index> (tableIndex, numVerts);
   }
};

Leveller::Leveller(UnstructuredGrid::const_ptr grid, const Scalar isovalue, Index processortype
         #ifdef CUTTINGSURFACE
            , int option
         #endif
            )
      : m_grid(grid)
   #ifdef CUTTINGSURFACE
      , m_option(option)
   #endif
      , m_isoValue(isovalue)
      , m_processortype(processortype)
      , gmin(std::numeric_limits<Scalar>::max())
      , gmax(-std::numeric_limits<Scalar>::max())
   {
      m_triangles = Triangles::ptr(new Triangles(Object::Initialized));
      m_triangles->setMeta(grid->meta());
   }

template<class Data, class pol>
Index Leveller::calculateSurface(Data &data) {

   thrust::counting_iterator<int> first(0);
   thrust::counting_iterator<int> last = first + m_grid->getNumElements();
   typedef thrust::tuple<typename Data::IndexIterator, typename Data::IndexIterator> Iteratortuple;
   typedef thrust::zip_iterator<Iteratortuple> ZipIterator;
   ZipIterator ElTupleVec(thrust::make_tuple(&data.m_el[0], &data.m_el[1]));
   data.m_ValidCellVector.resize(m_grid->getNumElements());
   typename Data::VectorIndexIterator end = thrust::copy_if(pol(), first, last, ElTupleVec, data.m_ValidCellVector.begin(), checkcell<Data>(data));
   size_t numValidCells = end-data.m_ValidCellVector.begin();
   data.m_caseNums.resize(numValidCells);
   data.m_numVertices.resize(numValidCells);
   data.m_LocationList.resize(numValidCells);
   thrust::transform(pol(), data.m_ValidCellVector.begin(), end, thrust::make_zip_iterator(thrust::make_tuple(data.m_caseNums.begin(),data.m_numVertices.begin())), classify_cell<Data>(data));
   thrust::exclusive_scan(pol(), data.m_numVertices.begin(), data.m_numVertices.end(), data.m_LocationList.begin());
   Index totalNumVertices = 0;
   if (!data.m_numVertices.empty())
      totalNumVertices += data.m_numVertices.back();
   if (!data.m_LocationList.empty())
      totalNumVertices += data.m_LocationList.back();
   for(int i = 0; i < data.m_numInVertData; i++){
      data.m_outVertData[i]->resize(totalNumVertices);
   }
   for(int i = 0; i < data.m_numInVertDataI; i++){
      data.m_outVertDataI[i]->resize(totalNumVertices);
   }
   for (int i=0; i<data.m_numInCellData; ++i) {
       data.m_outCellData[i]->resize(totalNumVertices/3);
   }
   for (int i=0; i<data.m_numInCellDataI; ++i) {
       data.m_outCellDataI[i]->resize(totalNumVertices/3);
   }
   thrust::counting_iterator<Index> start(0), finish(numValidCells);
   thrust::for_each(pol(), start, finish, process_Cell<Data>(data));
   return totalNumVertices;
}

bool Leveller::process() {
#ifndef CUTTINGSURFACE
   Vec<Scalar>::const_ptr dataobj = Vec<Scalar>::as(m_data);
   if (!dataobj)
      return false;
#else
#endif

   Index totalNumVertices = 0;

   switch (m_processortype) {

      case Host: {

         HostData HD(m_isoValue,
#ifndef CUTTINGSURFACE
               IsoDataFunctor(&dataobj->x()[0]),
#else
               IsoDataFunctor(vertex, point, direction, &m_grid->x()[0], &m_grid->y()[0], &m_grid->z()[0], m_option),
#endif
               m_grid->el(), m_grid->tl(), m_grid->cl(), m_grid->x(), m_grid->y(), m_grid->z());

         for (size_t i=0; i<m_vertexdata.size(); ++i) {
            if(Vec<Scalar,1>::const_ptr Scal = Vec<Scalar,1>::as(m_vertexdata[i])){
               HD.addmappeddata(Scal->x());
            }
            if(Vec<Scalar,3>::const_ptr Vect = Vec<Scalar,3>::as(m_vertexdata[i])){
               HD.addmappeddata(Vect->x());
               HD.addmappeddata(Vect->y());
               HD.addmappeddata(Vect->z());
            }
            if(Vec<Index,1>::const_ptr Idx = Vec<Index,1>::as(m_vertexdata[i])){
               HD.addmappeddata(Idx->x());
            }
         }
         for (size_t i=0; i<m_celldata.size(); ++i) {
            if(Vec<Scalar,1>::const_ptr Scal = Vec<Scalar,1>::as(m_celldata[i])){
               HD.addcelldata(Scal->x());
            }
            if(Vec<Scalar,3>::const_ptr Vect = Vec<Scalar,3>::as(m_celldata[i])){
               HD.addcelldata(Vect->x());
               HD.addcelldata(Vect->y());
               HD.addcelldata(Vect->z());
            }
            if(Vec<Index,1>::const_ptr Idx = Vec<Index,1>::as(m_celldata[i])){
               HD.addcelldata(Idx->x());
            }
         }

         totalNumVertices = calculateSurface<HostData, thrust::detail::host_t>(HD);

         {
             size_t idx=0;
             m_triangles->d()->x[0] = HD.m_outVertData[idx++];
             m_triangles->d()->x[1] = HD.m_outVertData[idx++];
             m_triangles->d()->x[2] = HD.m_outVertData[idx++];

             size_t idxI=0;
             for (size_t i=0; i<m_vertexdata.size(); ++i) {
                 if(Vec<Scalar>::as(m_vertexdata[i])){

                     Vec<Scalar,1>::ptr out = Vec<Scalar,1>::ptr(new Vec<Scalar,1>(Object::Initialized));
                     out->d()->x[0] = HD.m_outVertData[idx++];
                     out->setMeta(m_vertexdata[i]->meta());
                     out->setMapping(DataBase::Vertex);
                     m_outvertData.push_back(out);

                 }
                 if(Vec<Scalar,3>::as(m_vertexdata[i])){

                     Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(Object::Initialized));
                     out->d()->x[0] = HD.m_outVertData[idx++];
                     out->d()->x[1] = HD.m_outVertData[idx++];
                     out->d()->x[2] = HD.m_outVertData[idx++];
                     out->setMeta(m_vertexdata[i]->meta());
                     out->setMapping(DataBase::Vertex);
                     m_outvertData.push_back(out);

                 }
                 if(Vec<Index>::as(m_vertexdata[i])){

                     Vec<Index>::ptr out = Vec<Index>::ptr(new Vec<Index>(Object::Initialized));
                     out->d()->x[0] = HD.m_outVertDataI[idxI++];
                     out->setMeta(m_vertexdata[i]->meta());
                     out->setMapping(DataBase::Vertex);
                     m_outvertData.push_back(out);
                 }
             }
         }
         {
             size_t idx=0;
             size_t idxI=0;
             for (size_t i=0; i<m_celldata.size(); ++i) {
                 if(Vec<Scalar>::as(m_celldata[i])){

                     Vec<Scalar,1>::ptr out = Vec<Scalar,1>::ptr(new Vec<Scalar,1>(Object::Initialized));
                     out->d()->x[0] = HD.m_outCellData[idx++];
                     out->setMeta(m_celldata[i]->meta());
                     out->setMapping(DataBase::Element);
                     m_outcellData.push_back(out);

                 }
                 if(Vec<Scalar,3>::as(m_celldata[i])){

                     Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(Object::Initialized));
                     out->d()->x[0] = HD.m_outCellData[idx++];
                     out->d()->x[1] = HD.m_outCellData[idx++];
                     out->d()->x[2] = HD.m_outCellData[idx++];
                     out->setMeta(m_celldata[i]->meta());
                     out->setMapping(DataBase::Element);
                     m_outcellData.push_back(out);

                 }
                 if(Vec<Index>::as(m_celldata[i])){

                     Vec<Index>::ptr out = Vec<Index>::ptr(new Vec<Index>(Object::Initialized));
                     out->d()->x[0] = HD.m_outCellDataI[idxI++];
                     out->setMeta(m_celldata[i]->meta());
                     out->setMapping(DataBase::Element);
                     m_outcellData.push_back(out);
                 }
             }
         }
         break;
      }

      case Device: {

         DeviceData DD(m_isoValue,
#ifndef CUTTINGSURFACE
               IsoDataFunctor(&dataobj->x()[0]),
#else
               IsoDataFunctor(vertex, point, direction, &m_grid->x()[0], &m_grid->y()[0], &m_grid->z()[0], m_option),
#endif
               m_grid->getNumElements(), m_grid->el(), m_grid->tl(), m_grid->getNumCorners(), m_grid->cl(), m_grid->getSize(), m_grid->x(), m_grid->y(), m_grid->z());

#if 0
         totalNumVertices = calculateSurface<DeviceData, thrust::device>(DD);
#endif

         m_triangles->x().resize(totalNumVertices);
         Scalar *out_x = m_triangles->x().data();
         thrust::copy(DD.m_outVertData[0]->begin(), DD.m_outVertData[0]->end(), out_x);

         m_triangles->y().resize(totalNumVertices);
         Scalar *out_y = m_triangles->y().data();
         thrust::copy(DD.m_outVertData[1]->begin(), DD.m_outVertData[1]->end(), out_y);

         m_triangles->z().resize(totalNumVertices);
         Scalar *out_z = m_triangles->z().data();
         thrust::copy(DD.m_outVertData[2]->begin(), DD.m_outVertData[2]->end(), out_z);

         if(m_vertexdata.size()){
            if(Vec<Scalar>::as(m_vertexdata[0])){

               Vec<Scalar>::ptr out = Vec<Scalar>::ptr(new Vec<Scalar>(totalNumVertices));
               thrust::copy(DD.m_outVertData[3]->begin(), DD.m_outVertData[3]->end(), out->x().data());
               out->setMeta(m_vertexdata[0]->meta());
               m_outvertData.push_back(out);

            }
            if(Vec<Scalar,3>::as(m_vertexdata[0])){

               Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(totalNumVertices));
               thrust::copy(DD.m_outVertData[3]->begin(), DD.m_outVertData[3]->end(), out->x().data());
               thrust::copy(DD.m_outVertData[4]->begin(), DD.m_outVertData[4]->end(), out->y().data());
               thrust::copy(DD.m_outVertData[5]->begin(), DD.m_outVertData[5]->end(), out->z().data());
               out->setMeta(m_vertexdata[0]->meta());
               m_outvertData.push_back(out);

            }
         }
         break;
      }
   }

   return true;
}

#ifdef CUTTINGSURFACE
void Leveller::setCutData(const Vector m_vertex, const Vector m_point, const Vector m_direction){
   vertex = m_vertex;
   point = m_point;
   direction = m_direction;
}
#else    
void Leveller::setIsoData(Vec<Scalar>::const_ptr obj) {
   m_data = obj;
}
#endif

void Leveller::addMappedData(DataBase::const_ptr mapobj ){
    if (mapobj->mapping() == DataBase::Element)
        m_celldata.push_back(mapobj);
    else
        m_vertexdata.push_back(mapobj);
}

Object::ptr Leveller::result() {
      return m_triangles;
   }

DataBase::ptr Leveller::mapresult() const {
   if(m_outvertData.size())
      return m_outvertData[0];
   else if(m_outcellData.size())
      return m_outcellData[0];
   else
      return DataBase::ptr();
}

DataBase::ptr Leveller::cellresult() const {
   if(m_outcellData.size())
      return m_outcellData[0];
   else
      return DataBase::ptr();
}

std::pair<Scalar, Scalar> Leveller::range() {
   return std::make_pair(gmin, gmax);
}
