#include "hip/hip_runtime.h"
﻿//
//This code is used for both IsoCut and IsoSurface!
//

#include <sstream>
#include <iomanip>
#include <core/index.h>
#include <core/scalar.h>
#include <core/unstr.h>
#include <core/triangles.h>
#include <core/empty.h>
#include <core/shm.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include "tables.h"
#include "Leveller.h"

using namespace vistle;

#define lerp(a, b, t) ( a + t * (b - a) )

const Scalar EPSILON = 1.0e-10f;
const int maxNumdata = 6;

inline Scalar __host__ __device__ tinterp(Scalar iso, const Scalar &f0, const Scalar &f1) {

   Scalar diff = (f1 - f0);

   if (fabs(diff) < EPSILON)
      return 0;

   if (fabs(iso - f0) < EPSILON)
      return 0;

   if (fabs(iso - f1) < EPSILON)
      return 1;

   return (iso - f0) / diff;
}

#ifdef CUTTINGSURFACE

//! generate data on the fly for creating cutting surfaces as isosurfaces
struct IsoDataFunctor {

   IsoDataFunctor(const Vector &vertex, const Vector &point, const Vector &direction, const Scalar* x, const Scalar* y, const Scalar* z, int option)
      : m_x(x)
      , m_y(y)
      , m_z(z)
      , m_vertex(vertex)
      , m_point(point)
      , m_direction(direction)
      , m_distance(vertex.dot(point))
      , m_option(option)
      , m_vectorprod(m_direction.cross(m_point-m_vertex))
      , m_cylinderradius2(m_vectorprod.squaredNorm())
      , m_sphereradius2((m_vertex-m_point).squaredNorm())
   {}
   __host__ __device__ Scalar operator()(Index i) {

      switch(m_option) {
         case Plane: {
            Vector coordinates(m_x[i], m_y[i], m_z[i]);
            return m_vertex.dot(coordinates) - m_distance;
         }
         case Sphere: {
            Vector coordinates(m_x[i], m_y[i], m_z[i]);
            return coordinates.squaredNorm() - m_sphereradius2;
         }
         default: {
            // all cylinders
            Vector coordinates(m_x[i], m_y[i], m_z[i]);
            return (m_direction.cross(coordinates - m_vertex)).squaredNorm() - m_cylinderradius2;
         }
      }
   }
   const Scalar* m_x;
   const Scalar* m_y;
   const Scalar* m_z;
   const Vector m_vertex;
   const Vector m_point;
   const Vector m_direction;
   const Scalar m_distance;
   const int m_option;
   const Vector m_vectorprod;
   const Scalar m_cylinderradius2;
   const Scalar m_sphereradius2;
};

#else
//! fetch data from scalar field for generating isosurface
struct IsoDataFunctor {

   IsoDataFunctor(const Scalar *data)
      : m_volumedata(data)
   {}
   __host__ __device__ Scalar operator()(Index i) { return m_volumedata[i]; }

   const Scalar *m_volumedata;

};
#endif

struct HostData {

   Scalar m_isovalue;
   Index m_numinputdata;
   IsoDataFunctor m_isoFunc;
   const vistle::shm<Index>::array &m_el;
   const vistle::shm<Index>::array &m_cl;
   const vistle::shm<unsigned char>::array &m_tl;
   std::vector<Index> m_caseNums;
   std::vector<Index> m_numVertices;
   std::vector<Index> m_LocationList;
   std::vector<Index> m_ValidCellVector;
   const vistle::shm<Scalar>::array &m_x;
   const vistle::shm<Scalar>::array &m_y;
   const vistle::shm<Scalar>::array &m_z;
   std::vector<vistle::ShmVector<Scalar>::ptr> m_outData;
   std::vector<const Scalar*> m_inputpointer;
   std::vector<Scalar*> m_outputpointer;

   typedef vistle::shm<Index>::array::iterator IndexIterator;
   typedef std::vector<Index>::iterator VectorIndexIterator;

   HostData(Scalar isoValue
            , IsoDataFunctor isoFunc
            , const vistle::shm<Index>::array &el
            , const vistle::shm<unsigned char>::array &tl
            , const vistle::shm<Index>::array &cl
            , const vistle::shm<Scalar>::array &x
            , const vistle::shm<Scalar>::array &y
            , const vistle::shm<Scalar>::array &z
            )
      : m_isovalue(isoValue)
      , m_isoFunc(isoFunc)
      , m_el(el)
      , m_cl(cl)
      , m_tl(tl)
      , m_x(x)
      , m_y(y)
      , m_z(z)
   {
      m_inputpointer.push_back(x.data());
      m_inputpointer.push_back(y.data());
      m_inputpointer.push_back(z.data());

      for(size_t i = 0; i < m_inputpointer.size(); i++){
         m_outData.push_back(new vistle::ShmVector<Scalar>);
         m_outputpointer.push_back(NULL);
      }
      m_numinputdata = m_inputpointer.size();
   }

   void addmappeddata(const vistle::shm<Scalar>::array &mapdata){

      m_inputpointer.push_back(mapdata.data());
      m_outData.push_back(new vistle::ShmVector<Scalar>);
      m_outputpointer.push_back(NULL);
      m_numinputdata = m_inputpointer.size();

   }
};

struct DeviceData {

   Scalar m_isovalue;
   Index m_numinputdata;
   IsoDataFunctor m_isoFunc;
   thrust::device_vector<Index> m_el;
   thrust::device_vector<Index> m_cl;
   thrust::device_vector<unsigned char> m_tl;
   thrust::device_vector<Index> m_caseNums;
   thrust::device_vector<Index> m_numVertices;
   thrust::device_vector<Index> m_LocationList;
   thrust::device_vector<Index> m_ValidCellVector;
   thrust::device_vector<Scalar> m_x;
   thrust::device_vector<Scalar> m_y;
   thrust::device_vector<Scalar> m_z;
   std::vector<thrust::device_vector<Scalar> *> m_outData;
   std::vector<thrust::device_ptr<Scalar> > m_inputpointer;
   std::vector<thrust::device_ptr<Scalar> > m_outputpointer;
   typedef thrust::device_vector<Index>::iterator IndexIterator;

   DeviceData(Scalar isoValue
              , IsoDataFunctor isoFunc
              , const vistle::shm<Index>::array &el
              , const vistle::shm<unsigned char>::array &tl
              , const vistle::shm<Index>::array &cl
              , const vistle::shm<Scalar>::array &x
              , const vistle::shm<Scalar>::array &y
              , const vistle::shm<Scalar>::array &z)
   : m_isovalue(isoValue)
   , m_isoFunc(isoFunc)
   , m_el(el.begin(), el.end())
   , m_cl(cl.begin(), cl.end())
   , m_tl(tl.begin(), tl.end())
   , m_x(x.begin(), x.end())
   , m_y(y.begin(), y.end())
   , m_z(z.begin(), z.end())
   {
      m_inputpointer.push_back(m_x.data());
      m_inputpointer.push_back(m_y.data());
      m_inputpointer.push_back(m_z.data());

      for(size_t i = 0; i < m_inputpointer.size(); i++){
         m_outData.push_back(new thrust::device_vector<Scalar>);
      }
      m_outputpointer.resize(m_inputpointer.size());
      m_numinputdata = m_inputpointer.size();
   }
};

template<class Data>
struct process_Cell {
   process_Cell(Data &data) : m_data(data) {
      for (int i = 0; i < m_data.m_numinputdata; i++){
         m_data.m_outputpointer[i] = m_data.m_outData[i]->data();
      }
   }

   Data &m_data;

   __host__ __device__
   void operator()(Index ValidCellIndex) {

      const Index CellNr = m_data.m_ValidCellVector[ValidCellIndex];
      const Index Cellbegin = m_data.m_el[CellNr];
      const Index Cellend = m_data.m_el[CellNr+1];
      const Index numVert = m_data.m_numVertices[ValidCellIndex];
      const auto &cl = &m_data.m_cl[Cellbegin];

      switch (m_data.m_tl[CellNr]) {

         case UnstructuredGrid::HEXAHEDRON: {

            Scalar field[8];
            for (int idx = 0; idx < 8; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {

               const int edge = hexaTriTable[m_data.m_caseNums[ValidCellIndex]][idx];

               const int v1 = hexaEdgeTable[0][edge];
               const int v2 = hexaEdgeTable[1][edge];

               Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]);

               Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx;

               for(Index j = 0; j < m_data.m_numinputdata; j++){

                  m_data.m_outputpointer[j][outvertexindex] =
                     lerp(m_data.m_inputpointer[j][cl[v1]], m_data.m_inputpointer[j][cl[v2]], t);

               }
            }
            break;
         }

         case UnstructuredGrid::TETRAHEDRON: {

            Scalar field[4];
            for (int idx = 0; idx < 4; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {

               const int edge = tetraTriTable[m_data.m_caseNums[ValidCellIndex]][idx];

               const int v1 = tetraEdgeTable[0][edge];
               const int v2 = tetraEdgeTable[1][edge];

               Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]);

               Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx;

               for(Index j = 0; j < m_data.m_numinputdata; j++){

                  m_data.m_outputpointer[j][outvertexindex] =
                     lerp(m_data.m_inputpointer[j][cl[v1]], m_data.m_inputpointer[j][cl[v2]], t);

               }
            }
            break;
         }

         case UnstructuredGrid::PYRAMID: {

            Scalar field[5];
            for (int idx = 0; idx < 5; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {

               const int edge = pyrTriTable[m_data.m_caseNums[ValidCellIndex]][idx];

               const int v1 = pyrEdgeTable[0][edge];
               const int v2 = pyrEdgeTable[1][edge];

               Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]);

               Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx;

               for(Index j = 0; j < m_data.m_numinputdata; j++){

                  m_data.m_outputpointer[j][outvertexindex] =
                     lerp(m_data.m_inputpointer[j][cl[v1]], m_data.m_inputpointer[j][cl[v2]], t);

               }
            }
            break;
         }

         case UnstructuredGrid::PRISM: {

            Scalar field[6];
            for (int idx = 0; idx < 6; idx ++) {
               field[idx] = m_data.m_isoFunc(cl[idx]);
            }

            for (Index idx = 0; idx < m_data.m_numVertices[ValidCellIndex]; idx++) {

               const int edge = prismTriTable[m_data.m_caseNums[ValidCellIndex]][idx];

               const int v1 = prismEdgeTable[0][edge];
               const int v2 = prismEdgeTable[1][edge];

               Scalar t = tinterp(m_data.m_isovalue, field[v1], field[v2]);

               Index outvertexindex = m_data.m_LocationList[ValidCellIndex]+idx;

               for(Index j = 0; j < m_data.m_numinputdata; j++){

                  m_data.m_outputpointer[j][outvertexindex] =
                     lerp(m_data.m_inputpointer[j][cl[v1]], m_data.m_inputpointer[j][cl[v2]], t);

               }
            }
            break;
         }

         case UnstructuredGrid::POLYHEDRON: {

            Index sidebegin = InvalidIndex;
            bool vertexSaved = false;
            Scalar savedData [maxNumdata];
            Index j = 0;
            int flag = 0;
            Scalar middleData[maxNumdata];
            for(int i = 0; i < maxNumdata; i++ ){
               middleData[i] = 0;
            };
            Scalar cd1 [maxNumdata];
            Scalar cd2 [maxNumdata];

            Index outIdx = m_data.m_LocationList[ValidCellIndex];
            for (Index i = Cellbegin; i < Cellend; i++) {

               const Index c1 = m_data.m_cl[i];
               const Index c2 = m_data.m_cl[i+1];

               if (c1 == sidebegin) {

                  sidebegin = InvalidIndex;
                  if (vertexSaved) {

                     for(Index i = 0; i < m_data.m_numinputdata; i++){
                        m_data.m_outputpointer[i][outIdx] = savedData[i];
                     };

                     outIdx += 2;
                     vertexSaved=false;
                  }
                  continue;
               } else if(sidebegin == InvalidIndex) { //Wenn die Neue Seite beginnt

                  flag = 0;
                  sidebegin = c1;
                  vertexSaved = false;
               }

               for(int i = 0; i < m_data.m_numinputdata; i++){
                  cd1[i] = m_data.m_inputpointer[i][c1];
                  cd2[i] = m_data.m_inputpointer[i][c2];
               }

               Scalar d1 = m_data.m_isoFunc(c1);
               Scalar d2 = m_data.m_isoFunc(c2);
               Scalar t = tinterp(m_data.m_isovalue, d1, d2);

               if (d1 <= m_data.m_isovalue && d2 > m_data.m_isovalue) {
                  Scalar v [maxNumdata];
                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     v[i] = lerp(cd1[i], cd2[i], t);
                  };

                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     middleData[i] += v[i];
                  }

                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     m_data.m_outputpointer[i][outIdx] = v[i];
                  }

                  ++outIdx;
                  ++j;
                  flag = 1;

               } else if (d1 > m_data.m_isovalue && d2 <= m_data.m_isovalue) {

                  Scalar v [maxNumdata];

                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     v[i] = lerp(cd1[i], cd2[i], t);
                  };
                  for(Index i = 0; i < m_data.m_numinputdata; i++){
                     middleData[i] += v[i];
                  };
                  ++j;
                  if (flag == 1) { //fall 2 nach fall 1
                     for(Index i = 0; i < m_data.m_numinputdata; i++){
                        m_data.m_outputpointer[i][outIdx] = v[i];
                     }
                     outIdx += 2;
                  } else { //fall 2 zuerst

                     for(Index i = 0; i < m_data.m_numinputdata; i++){
                        savedData[i] = v[i];
                     }
                     vertexSaved=true;
                  }
               }
            }
            for(Index i = 0; i < m_data.m_numinputdata; i++){
               middleData[i] /= j;
            };
            for (Index i = 2; i < numVert; i += 3) {
               const Index idx = m_data.m_LocationList[ValidCellIndex]+i;
               for(Index i = 0; i < m_data.m_numinputdata; i++){
                  m_data.m_outputpointer[i][idx] = middleData[i];
               }
            };
            break;
         }
      }
   }
};

template<class Data>
struct checkcell {

   typedef float argument_type;
   typedef float result_type;
   Data &m_data;
   checkcell(Data &data) : m_data(data) {}

   __host__ __device__ int operator()(const thrust::tuple<Index,Index> iCell) const {

      int havelower = 0;
      int havehigher = 0;
      Index Cell = iCell.get<0>();
      Index nextCell = iCell.get<1>();
      for (Index i=Cell; i<nextCell; i++) {
         float val = m_data.m_isoFunc(m_data.m_cl[i]);
         if (val>m_data.m_isovalue) {
            havelower=1;
            if (havehigher)
               return 1;
         } else {
            havehigher=1;
            if (havelower)
               return 1;
         }
      }
      return 0;
   }
};

template<class Data>
struct classify_cell {

   classify_cell(Data &data) : m_data(data) {}

   Data &m_data;

   __host__ __device__ thrust::tuple<Index,Index> operator()(Index CellNr) {

      uint tableIndex = 0;
      Index Start = m_data.m_el[CellNr];
      Index diff = m_data.m_el[CellNr+1]-Start;
      unsigned char CellType = m_data.m_tl[CellNr];
      int numVerts = 0;
      if (CellType != UnstructuredGrid::POLYHEDRON) {
         for (int idx = 0; idx < diff; idx ++) {
            tableIndex += (((int) (m_data.m_isoFunc(m_data.m_cl[Start+idx]) > m_data.m_isovalue)) << idx);
         }
      }
      switch (CellType) {

         case UnstructuredGrid::HEXAHEDRON:
            numVerts = hexaNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::TETRAHEDRON:
            numVerts = tetraNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PYRAMID:
            numVerts = pyrNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::PRISM:
            numVerts = prismNumVertsTable[tableIndex];
            break;

         case UnstructuredGrid::POLYHEDRON: {

            Index sidebegin = InvalidIndex;
            Index vertcounter = 0;
            for (Index i = Start; i < Start + diff; i++) {

               if (m_data.m_cl[i] == sidebegin) {
                  sidebegin = InvalidIndex;
                  continue;
               }

               if (sidebegin == InvalidIndex) {
                  sidebegin = m_data.m_cl[i];
               }

               if (m_data.m_isoFunc(m_data.m_cl[i]) <= m_data.m_isovalue && m_data.m_isoFunc(m_data.m_cl[i+1]) > m_data.m_isovalue) {

                  vertcounter += 1;
               } else if(m_data.m_isoFunc(m_data.m_cl[i]) > m_data.m_isovalue && m_data.m_isoFunc(m_data.m_cl[i+1]) <= m_data.m_isovalue) {

                  vertcounter += 1;
               }
            }
            numVerts = vertcounter + vertcounter/2;
            break;
         }
      };
      return thrust::make_tuple<Index, Index> (tableIndex, numVerts);
   }
};

Leveller::Leveller(UnstructuredGrid::const_ptr grid, const Scalar isovalue, Index processortype
         #ifdef CUTTINGSURFACE
            , int option
         #endif
            )
      : m_grid(grid)
   #ifdef CUTTINGSURFACE
      , m_option(option)
   #endif
      , m_isoValue(isovalue)
      , m_processortype(processortype)
      , gmin(std::numeric_limits<Scalar>::max())
      , gmax(-std::numeric_limits<Scalar>::max())
   {
      m_triangles = Triangles::ptr(new Triangles(Object::Initialized));
      m_triangles->setMeta(grid->meta());
   }

template<class Data, class pol>
Index Leveller::calculateSurface(Data &data) {

   thrust::counting_iterator<int> first(0);
   thrust::counting_iterator<int> last = first + m_grid->getNumElements();
   typedef thrust::tuple<typename Data::IndexIterator, typename Data::IndexIterator> Iteratortuple;
   typedef thrust::zip_iterator<Iteratortuple> ZipIterator;
   ZipIterator ElTupleVec(thrust::make_tuple(data.m_el.begin(), data.m_el.begin()+1));
   data.m_ValidCellVector.resize(m_grid->getNumElements());
   typename Data::VectorIndexIterator end = thrust::copy_if(pol(), first, last, ElTupleVec, data.m_ValidCellVector.begin(), checkcell<Data>(data));
   size_t numValidCells = end-data.m_ValidCellVector.begin();
   data.m_caseNums.resize(numValidCells);
   data.m_numVertices.resize(numValidCells);
   data.m_LocationList.resize(numValidCells);
   thrust::transform(pol(), data.m_ValidCellVector.begin(), end, thrust::make_zip_iterator(thrust::make_tuple(data.m_caseNums.begin(),data.m_numVertices.begin())), classify_cell<Data>(data));
   thrust::exclusive_scan(pol(), data.m_numVertices.begin(), data.m_numVertices.end(), data.m_LocationList.begin());
   Index totalNumVertices = 0;
   if (!data.m_numVertices.empty())
      totalNumVertices += data.m_numVertices.back();
   if (!data.m_LocationList.empty())
      totalNumVertices += data.m_LocationList.back();
   for(int i = 0; i < data.m_numinputdata; i++){
      data.m_outData[i]->resize(totalNumVertices);
   };
   thrust::counting_iterator<Index> start(0), finish(numValidCells);
   thrust::for_each(pol(), start, finish, process_Cell<Data>(data));
   return totalNumVertices;
}

bool Leveller::process() {
   if(m_mapdata.size()){
      Vec<Scalar>::const_ptr mapdataobj = Vec<Scalar>::as(m_mapdata[0]);
   }
#ifndef CUTTINGSURFACE
   Vec<Scalar>::const_ptr dataobj = Vec<Scalar>::as(m_data);
   if (!dataobj)
      return false;
#else
#endif

   Index totalNumVertices = 0;

   switch (m_processortype) {

      case Host: {

         HostData HD(m_isoValue,
#ifndef CUTTINGSURFACE
               IsoDataFunctor(dataobj->x().data()),
#else
               IsoDataFunctor(vertex, point, direction, m_grid->x().data(), m_grid->y().data(), m_grid->z().data(), m_option),
#endif
               m_grid->el(), m_grid->tl(), m_grid->cl(), m_grid->x(), m_grid->y(), m_grid->z());

         if(m_mapdata.size()){
            if(Vec<Scalar,1>::const_ptr Scal = Vec<Scalar,1>::as(m_mapdata[0])){
               HD.addmappeddata(Scal->x());
            }
            if(Vec<Scalar,3>::const_ptr Vect = Vec<Scalar,3>::as(m_mapdata[0])){
               HD.addmappeddata(Vect->x());
               HD.addmappeddata(Vect->y());
               HD.addmappeddata(Vect->z());
            }

         }

         totalNumVertices = calculateSurface<HostData, thrust::detail::host_t>(HD);

         m_triangles->d()->x[0] = HD.m_outData[0];
         m_triangles->d()->x[1] = HD.m_outData[1];
         m_triangles->d()->x[2] = HD.m_outData[2];

         if(m_mapdata.size()){
            if(Vec<Scalar>::as(m_mapdata[0])){

               Vec<Scalar,1>::ptr out = Vec<Scalar,1>::ptr(new Vec<Scalar,1>(Object::Initialized));
               out->d()->x[0] = HD.m_outData[3];
               m_outmapData.push_back(out);

            }
            if(Vec<Scalar,3>::as(m_mapdata[0])){

               Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(Object::Initialized));
               out->d()->x[0] = HD.m_outData[3];
               out->d()->x[1] = HD.m_outData[4];
               out->d()->x[2] = HD.m_outData[5];
               m_outmapData.push_back(out);

            }

            m_outmapData.back()->setMeta(m_mapdata[0]->meta());
         }
         break;
      }

      case Device: {

         DeviceData DD(m_isoValue,
#ifndef CUTTINGSURFACE
               IsoDataFunctor(dataobj->x().data()),
#else
               IsoDataFunctor(vertex, point, direction, m_grid->x().data(), m_grid->y().data(), m_grid->z().data(), m_option),
#endif
               m_grid->el(), m_grid->tl(), m_grid->cl(), m_grid->x(), m_grid->y(), m_grid->z());

#if 0
         totalNumVertices = calculateSurface<DeviceData, thrust::device>(DD);
#endif

         m_triangles->x().resize(totalNumVertices);
         Scalar *out_x = m_triangles->x().data();
         thrust::copy(DD.m_outData[0]->begin(), DD.m_outData[0]->end(), out_x);

         m_triangles->y().resize(totalNumVertices);
         Scalar *out_y = m_triangles->y().data();
         thrust::copy(DD.m_outData[1]->begin(), DD.m_outData[1]->end(), out_y);

         m_triangles->z().resize(totalNumVertices);
         Scalar *out_z = m_triangles->z().data();
         thrust::copy(DD.m_outData[2]->begin(), DD.m_outData[2]->end(), out_z);

         if(m_mapdata.size()){
            if(Vec<Scalar>::as(m_mapdata[0])){

               Vec<Scalar>::ptr out = Vec<Scalar>::ptr(new Vec<Scalar>(totalNumVertices));
               thrust::copy(DD.m_outData[3]->begin(), DD.m_outData[3]->end(), out->x().data());
               m_outmapData.push_back(out);

            }
            if(Vec<Scalar,3>::as(m_mapdata[0])){

               Vec<Scalar,3>::ptr out = Vec<Scalar,3>::ptr(new Vec<Scalar,3>(totalNumVertices));
               thrust::copy(DD.m_outData[3]->begin(), DD.m_outData[3]->end(), out->x().data());
               thrust::copy(DD.m_outData[4]->begin(), DD.m_outData[4]->end(), out->y().data());
               thrust::copy(DD.m_outData[5]->begin(), DD.m_outData[5]->end(), out->z().data());
               m_outmapData.push_back(out);

            }

            m_outmapData.back()->setMeta(m_mapdata[0]->meta());
         }
         break;
      }
   }
   m_triangles->cl().resize(totalNumVertices);
   Index *out_cl = m_triangles->cl().data();
   thrust::counting_iterator<Index> first(0), last(totalNumVertices);
   thrust::copy(first, last, out_cl);

   return true;
}

#ifdef CUTTINGSURFACE
void Leveller::setCutData(const Vector m_vertex, const Vector m_point, const Vector m_direction){
   vertex = m_vertex;
   point = m_point;
   direction = m_direction;
}
#else    
void Leveller::setIsoData(Vec<Scalar>::const_ptr obj) {
   m_data = obj;
}
#endif

void Leveller::addMappedData(Object::const_ptr mapobj ){
   m_mapdata.push_back(mapobj);
}

Object::ptr Leveller::result() {
      return m_triangles;
   }

Object::ptr Leveller::mapresult() {
   if(m_outmapData.size())
      return m_outmapData[0];
   else
      return Object::ptr(new Empty(Object::Initialized));
}

std::pair<Scalar, Scalar> Leveller::range() {
   return std::make_pair(gmin, gmax);
}
